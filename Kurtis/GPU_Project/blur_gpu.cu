#include "hip/hip_runtime.h"
#include "myBlur_gpu.h"

int padWidth;
int padHeight;

__global__ void myBlur_gpu_kernal(InputArray src, OutputArray dst, Size ksize) {
	//get the index for a given block and thread, (blockDim = threads per block)
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < ksize.width && y < ksize.height) return;

	x += padWidth;
	y += padHeight;
	uchar3 sum = 0;

	//calculate the summation of the surounding pixels
	for (int i = x-padWidth; i <= x+padWidth; i++) {
		for (int j = y-padHeight; j <= y+padHeight; j++) {
				 sum += src(j, i);
		}
	}

	//set pixel as the average of the surounding pixels
	dst(y, x) = sum / (ksize.width * ksize.height);
}

void myBlur_gpu(InputArray src, OutputArray dst, Size ksize) {
	if (ksize.width == 0 || ksize.height == 0) return;

	//divide the array into blocks
	const int threadsPerBlock = 32;
	const dim3 gridSize(ceil((float) src.cols / threadsPerBlock), ceil((float) src.rows / threadsPerBlock), 1);
  	const dim3 blockSize(threadsPerBlock, threadsPerBlock, 1);

  	//allocate space for dst
  	dst.create(src.size(), src.type());

  	//pad image
  	padWidth = ceil((ksize.width - 1) / 2.0);
  	padHeight = ceil((ksize.height - 1) / 2.0);

  	InputArray _src();
  	_src.create(src.size(), src.type());
  	cv::copyMakeBorder(src, _src, padHeight, padHeight, padWidth, padWidth, BORDER_REPLICATE);

  	//apply average blur
	myBlur_gpu_kernal<<<gridSize, blockSize>>>(_src, dst, ksize);
}